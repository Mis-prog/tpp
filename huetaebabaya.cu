#include "hip/hip_runtime.h"

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define N 10000 // Total threads
#define M 1024 // Threads per block

// Host - PC
// Device - GPU

void random_floats(float *x, int n, bool printRes);
void check_results(float *x, float *y, float *z, int n);

__global__ void add(float *a, float *b, float *c)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < N) { c[index] = a[index] + b[index]; }
}

int main(void) 
{	
	srand(time(NULL));
	float *a = new float[N], *b = new float[N], *c = new float[N];
	float *a1, *b1, *c1;
	int size = sizeof(float) * N;
	bool printResults = false;
	
	printf("Size: %d\n", N);
	// Allocate space (without unified memory)
	hipMalloc((void**)&a1, size);
	hipMalloc((void**)&b1, size);
	hipMalloc((void**)&c1, size);		

	// Initialize arrays
	random_floats(a, N, printResults);
	random_floats(b, N, printResults);
			
	hipEvent_t startHTD, stopHTD;
	float timeHTD = 0;
	hipEventCreateWithFlags(&startHTD, 0);
	hipEventCreateWithFlags(&stopHTD, 0);
	hipEventRecord(startHTD, 0);
	// copying data from host to device
	hipMemcpy(a1, a, size, hipMemcpyHostToDevice);	
	hipMemcpy(b1, b, size, hipMemcpyHostToDevice);	
//	cudaMemcpy(c1, c, size, cudaMemcpyHostToDevice);	
	hipEventRecord(stopHTD, 0);
	hipEventSynchronize(stopHTD);
	hipEventElapsedTime(&timeHTD, startHTD, stopHTD);
	printf("Elapsed time from Host to Device: %.3f ms\n", timeHTD);
	hipEventDestroy(startHTD);
	hipEventDestroy(stopHTD);	
	
	// timings
	hipEvent_t start, stop;
	float time = 0;
	hipEventCreateWithFlags(&start, 0);
	hipEventCreateWithFlags(&stop, 0);
	hipEventRecord(start, 0);

		
	add <<< (N + M - 1) / M, M>>> (a1, b1, c1); // Launch add() kernel on GPU
	
	// (N + M - 1 / M) - universal formula for the amount of blocks	
	// M - amount of threads
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Elapsed time : %.3f ms\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	// timings
	
	hipEvent_t startDTH, stopDTH;
	float timeDTH = 0;
	hipEventCreateWithFlags(&startDTH, 0);
	hipEventCreateWithFlags(&stopDTH, 0);
	hipEventRecord(startDTH, 0);
	// copying data from device to host
//	cudaMemcpy(a, a1, size, cudaMemcpyDeviceToHost);	
//	cudaMemcpy(b, b1, size, cudaMemcpyDeviceToHost);	
	hipMemcpy(c, c1, size, hipMemcpyDeviceToHost);
	hipEventRecord(stopDTH, 0);
	hipEventSynchronize(stopDTH);
	hipEventElapsedTime(&timeDTH, startDTH, stopDTH);
	printf("Elapsed time from Device to Host: %.3f ms\n", timeDTH);
	hipEventDestroy(startDTH);
	hipEventDestroy(stopDTH);
	
	hipFree(a1);
	hipFree(b1);
	hipFree(c1);
	
	if (printResults)
	{
		check_results(a, b, c, N);
	}
	
	// Cleanup
	hipFree(a);
	hipFree(b);
	hipFree(c);
	delete[] a;
	delete[] b;
	delete[] c;
	return 0;
}

inline void check_cuda_errors(const char *filename, const int line_number) 
{
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error != hipSuccess) 
	{
		printf("CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString(error));
		exit(-1);
	}
}

void random_floats(float *x, int n, bool printValues)
{
	for (int i = 0; i < n; i++)
	{
		x[i] = abs(sin(rand()) * 100);
		if (printValues)
		{
			printf("|sin(i)| * 100 = %f\n", x[i]);
		}
	}
} 

void check_results(float *x, float *y, float *z, int n)
{
	for (int i = 0; i < n; i++)
	{
		printf("%d", i);
		if (abs(x[i] + y[i] - z[i]) < 0.001)
		{
			printf("abs(%f + %f - %f) = %f\n", x[i], y[i], z[i], x[i]+y[i]-z[i]);
			printf("Valid result\n");
		}
		else
		{
			printf("abs(%f + %f - %f) != %f\n", x[i], y[i], z[i], x[i]+y[i]-z[i]);
			printf("Invalid result\n");
		}
	}
}
